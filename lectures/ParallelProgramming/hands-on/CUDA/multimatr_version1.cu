#include "hip/hip_runtime.h"
#include<stdio.h>
#include<assert.h>

#define N 10 //0000
#define NUM_THREADS 512

__global__ void matmult( double * d_A, double * d_B, double * d_C, int SIZE )
{
int i = threadIdx.x + (blockIdx.x * blockDim.x);
int j = threadIdx.y + (blockIdx.y * blockDim.y);

for (i=0; i<SIZE; i++){
for (j=0; j<SIZE; j++){
for (int k=0; k<SIZE; k++){
d_C[j+i*SIZE]+=d_A[i*SIZE+k]*d_B[k*SIZE+j];
}
}
}

}

void PRINT_MAT(int N, int M, double * matr){
  for(int j = 0; j < N; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%f ",matr[i+j*M]);
    }
    printf("\n");
  }
}

}

int main(){

double h_A ,h_B, h_C;
double d_A ,d_B, d_C;
size_t matsize = N * N * sizeof(double); //long integer

h_A = (double *) malloc( matsize );
h_B = (double *) malloc( matsize );
h_C = (double *) malloc( matsize );

hipMalloc((void**), &d_A, matsize );
hipMalloc((void**), &d_B, matsize );
hipMalloc((void**), &d_C, matsize );

for(int i=0;i<N;i++){
h_A[i]=(double *)i;
h_B[i]=(double *)i;
h_C[i]=0.;
}

PRINT_MAT(N,N,h_A);
PRINT_MAT(N,N,h_B);

hipMemcpy( d_A, h_A, matsize, hipMemcpyHostToDevice );
hipMemcpy( d_B, h_B, matsize, hipMemcpyHostToDevice );
hipMemcpy( d_C, h_C, matsize, hipMemcpyHostToDevice );

matmult<<< ((N*N)+NUM_THREADS)/NUM_THREADS , NUM_THREADS >>>( d_A, d_B, d_C, N )
hipMemcpy( h_C, d_C, matsize, hipMemcpyDeviceToHost );

PRINT_MAT(N,N,h_C);

free(h_A);
free(h_B);
free(h_C);

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

return 0;

}