#include "hip/hip_runtime.h"
#include<stdio.h>
#include<assert.h>

#define N 4096
//#define NUM_THREADS 512
#define BK_SIZE 512

__global__ void matmult( double * d_A, double * d_B, double * d_C )
{
  __shared__ double Arow[N];
  
  int idx = threadIdx.x;
  int idy = blockIdx.y;
  while(idy<N){
    Arow[idy]=d_A[idy+N*idx];
    idy+=blockDim.x;
  }
  idy+= threadIdx.y;
  
  for (int k=0; k<N; k++){
    d_C[idx*N+idy]+=Arow[k]*d_B[k*N+idy];
    idx+=threadDim.x;
  }
   
  __syncthreads();
  
}

void PRINT_MAT(int P, int M, double * matr){
  for(int j = 0; j < P; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%f ",matr[i+j*M]);
     }
    printf("\n");
  }
}

}

int main(){
  
  double * h_A , * h_B, * h_C;
  double * d_A , * d_B, * d_C;
  size_t matsize = N * N * sizeof(double); //long integer
  
  h_A = (double *) malloc( matsize );
  h_B = (double *) malloc( matsize );
  h_C = (double *) malloc( matsize );
  
  hipMalloc((void**) &d_A, matsize );
  hipMalloc((void**) &d_B, matsize );
  hipMalloc((void**) &d_C, matsize );
  
  for(int i=0;i<N;i++){
    h_A[i]=(double )i;
    h_B[i]=(double )i;
    h_C[i]=0.;
  }
  
  PRINT_MAT(N,N,h_A);
  PRINT_MAT(N,N,h_B);
  
  hipMemcpy( d_A, h_A, matsize, hipMemcpyHostToDevice );
  hipMemcpy( d_B, h_B, matsize, hipMemcpyHostToDevice );
  hipMemcpy( d_C, h_C, matsize, hipMemcpyHostToDevice );

  dim3 blockDim(BK_SIZE, BK_SIZE);
  dim3 gridDim(N/BK_SIZE, N/BK_SIZE);
  matmult<<< gridDim, blockDim >>>( d_A, d_B, d_C);
  hipMemcpy( h_C, d_C, matsize, hipMemcpyDeviceToHost );
  
  PRINT_MAT(N,N,h_C);
  
  free(h_A);
  free(h_B);
  free(h_C);
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  
  return 0;
  
}
