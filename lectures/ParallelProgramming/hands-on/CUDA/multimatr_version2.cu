#include "hip/hip_runtime.h"
#include<stdio.h>
#include<assert.h>

#define N 10 //0000
#define NUM_THREADS 512

__global__ void matmult( double * d_A, double * d_B, double * d_C, int SIZE )
{
__shared__ double Arow[SIZE];

//int idx = threadIdx.x + (blockIdx.x * blockDim.x);
//int idy = threadIdx.y + (blockIdx.y * blockDim.y);

for (int i=0;i<SIZE;i++){
Arow[i]=d_A[i+SIZE*blockIdx.x];
}

for (int i=0; i<SIZE; i++){
for (int j=0; j<SIZE; j++){
d_C[i]+=Arow[j]*d_B[j*SIZE+i];
}
}
__syncthreads();

}

void PRINT_MAT(int N, int M, double * matr){
  for(int j = 0; j < N; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%f ",matr[i+j*M]);
    }
    printf("\n");
  }
}

}

int main(){

double h_A ,h_B, h_C;
double d_A ,d_B, d_C;
size_t matsize = N * N * sizeof(double); //long integer

h_A = (double *) malloc( matsize );
h_B = (double *) malloc( matsize );
h_C = (double *) malloc( matsize );

hipMalloc((void**), &d_A, matsize );
hipMalloc((void**), &d_B, matsize );
hipMalloc((void**), &d_C, matsize );

for(int i=0;i<N;i++){
h_A[i]=(double *)i;
h_B[i]=(double *)i;
h_C[i]=0.;
}

PRINT_MAT(N,N,h_A);
PRINT_MAT(N,N,h_B);

hipMemcpy( d_A, h_A, matsize, hipMemcpyHostToDevice );
hipMemcpy( d_B, h_B, matsize, hipMemcpyHostToDevice );
hipMemcpy( d_C, h_C, matsize, hipMemcpyHostToDevice );

matmult<<< ((N*N)+NUM_THREADS)/NUM_THREADS , NUM_THREADS >>>( d_A, d_B, d_C, N )
hipMemcpy( h_C, d_C, matsize, hipMemcpyDeviceToHost );

PRINT_MAT(N,N,h_C);

free(h_A);
free(h_B);
free(h_C);

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

return 0;

}