
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<assert.h>

//cudaMemcpy( dest, source, sizeinbytes, cudaMemcpyHostToDevice | cudaMemcpyDeviceToHost );
//cudaMalloc( (void **) &my_ptr, sizeinbytes );

#define SIZE 12 //0000000
#define NUM_THREADS 512

__global__ void add( int * d_a, int * d_b, int * d_c )
{
  int idx = threadIdx.x + (blockIdx.x * blockDim.x );
  if( idx < SIZE )
    d_c[idx] = d_a[idx] + d_b[idx];
}


int main()
{
  int * h_a, * h_b, * h_c;
  int * d_a, * d_b, * d_c;
  int i;
  size_t size_in_bytes = SIZE * sizeof(int);

  h_a = (int *) malloc( size_in_bytes );
  h_b = (int *) malloc( size_in_bytes );
  h_c = (int *) malloc( size_in_bytes );

  hipMalloc( (void **) &d_a, size_in_bytes );
  hipMalloc( (void **) &d_b, size_in_bytes );
  hipMalloc( (void **) &d_c, size_in_bytes );

  for( i = 0; i < SIZE; i++ ){
    h_a[i] = 1;
    h_b[i] = 2;
  }

  hipMemcpy( d_a, h_a, size_in_bytes, hipMemcpyHostToDevice );
  hipMemcpy( d_b, h_b, size_in_bytes, hipMemcpyHostToDevice );

  add<<< ( SIZE + NUM_THREADS ) / NUM_THREADS, NUM_THREADS >>>( d_a, d_b, d_c );

  hipMemcpy( h_c, d_c, size_in_bytes, hipMemcpyDeviceToHost );  

  if( SIZE < 100 ){
    for( i = 0; i < SIZE; i++)
      fprintf( stdout, " %d", h_c[i] );
  }
  
  free( h_c );
  free( h_b );
  free( h_a );

  hipFree( d_a );
  hipFree( d_b );
  hipFree( d_c );

  return 0;

}
