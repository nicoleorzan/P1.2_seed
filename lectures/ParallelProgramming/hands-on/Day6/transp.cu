#include "hip/hip_runtime.h"
#include<stdio.h>
#include<assert.h>

#define N 10 //0000
#define NUM_THREADS 512

__global__ void transpose( double * d_A, double * d_AT )
{
int idx = threadIdx.x + (blockIdx.x * blockDim.x);
int idy = threadIdx.y + (blockIdx.y * blockDim.y);

if (idx<N){
if (idy<N){
d_AT[idx + idy*N] = d_A[ind*N + idy];
}
}

void PRINT_MAT(int N, int M, double * matr){
  for(int j = 0; j < N; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%f ",matr[i+j*M]);
    }
    printf("\n");
  }
}

}

int main(){

double h_A, h_AT;
double d_A, d_AT;
size_t matsize = N * N * sizeof(double); //long integer

h_A = (double *) malloc( matsize );
h_AT = (double *) malloc( matsize );

hipMalloc((void**), &d_A, matsize );
hipMalloc((void**), &d_AT, matsize );

for(int i=0;i<N;i++){
h_A[i]=(double *)i;
h_AT[i]=0.;
}

PRINT_MAT(N,N,h_A);

hipMemcpy( d_A, h_A, matsize, hipMemcpyHostToDevice );
hipMemcpy( d_AT, h_AT, matsize, hipMemcpyHostToDevice );

transpose<<< ((N*N)+NUM_THREADS)/NUM_THREAD , NUM_THREADS >>>( d_A , d_AT )
hipMemcpy( h_AT, d_AT, matsize, hipMemcpyDeviceToHost );

PRINT_MAT(N,N,h_AT);

free(h_A);
free(h_AT);

hipFree(d_A);
hipFree(d_AT);

return 0;

}
