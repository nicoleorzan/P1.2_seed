
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

#define NUMTHREADSPERBLOCK 16

/*** function declarations ***/

void PRINT_MAT(int P, int M, double * matr);

// save matrix to file
void save_gnuplot( double *M, size_t dim );

// evolve Jacobi
__global__  void evolve( double * matrix, double *matrix_new, int dimension );


// return the elapsed time
double seconds( void );

/*** end function declaration ***/




int main(int argc, char* argv[]){

  // timing variables
  double t_start, t_end, increment;

  // indexes for loops
  size_t i, j, it;

  // initialize matrix

  double *matrix, *matrix_new, *tmp_matrix;
  double *d_matrix, *d_matrix_new;
  int dimension = 0, iterations = 0, row_peek = 0, col_peek = 0;
  size_t byte_dimension = 0;

 // check on input parameters

  if(argc != 5) {
    fprintf(stderr,"\nwrong number of arguments. Usage: ./a.out dim it n m\n");
    return 1;
  }

  dimension = atoi(argv[1]);
  iterations = atoi(argv[2]);
  row_peek = atoi(argv[3]);
  col_peek = atoi(argv[4]);

  printf("matrix size = %zu\n", dimension);
  printf("number of iterations = %zu\n", iterations);
  printf("element for checking = Mat[%zu,%zu]\n",row_peek, col_peek);


  if((row_peek > dimension) || (col_peek > dimension)){
    fprintf(stderr, "Cannot Peek a matrix element outside of the matrix dimension\n");
    fprintf(stderr, "Arguments n and m must be smaller than %zu\n", dimension);
    return 1;
  }


  byte_dimension = sizeof(double*) * ( dimension + 2 ) * ( dimension + 2 );
  matrix = ( double* )malloc( byte_dimension );
  matrix_new = ( double* )malloc( byte_dimension );

  memset( matrix, 0, byte_dimension );
  memset( matrix_new, 0, byte_dimension );

  hipMalloc((void**) &d_matrix, byte_dimension ); //allocating space for d_matrix
  hipMalloc((void**) &d_matrix_new, byte_dimension );

  //fill initial values  

  for( i = 1; i <= dimension; ++i ){
    for( j = 1; j <= dimension; ++j ){
      matrix[ ( i * ( dimension + 2 ) ) + j ] = 0.5;
    }
  }
	     
  // set up borders 
  increment = 100.0 / ( dimension + 1 );

  
 for( i=1; i <= dimension+1; ++i ){
    matrix[ i * ( dimension + 2 ) ] = i * increment;
    matrix[ ( ( dimension + 1 ) * ( dimension + 2 ) ) + ( dimension + 1 - i ) ] = i * increment;
    matrix_new[ i * ( dimension + 2 ) ] = i * increment;
    matrix_new[ ( ( dimension + 1 ) * ( dimension + 2 ) ) + ( dimension + 1 - i ) ] = i * increment;
  }

  printf("initial matrix\n");
  PRINT_MAT(dimension+2,dimension+2,matrix);

  // start algorithm
  t_start = seconds();

  hipMemcpy( d_matrix, matrix, byte_dimension, hipMemcpyHostToDevice );
  hipMemcpy( d_matrix_new, matrix_new, byte_dimension, hipMemcpyHostToDevice );

  //dim3 gridDim( (dimension+NUMTHREADSPERBLOCK)/NUMTHREADSPERBLOCK, (dimension+NUMTHREADSPERBLOCK)/NUMTHREADSPERBLOCK );
  dim3 gridDim( (NUMTHREADSPERBLOCK-2)*dimension + 2*NUMTHREADSPERBLOCK, (NUMTHREADSPERBLOCK-2)*dimension + 2*NUMTHREADSPERBLOCK );
  //dim3 gridDim( (dimension+2)/(NUMTHREADSPERBLOCK) +2 , (dimension+2)/(NUMTHREADSPERBLOCK) +2 );
  //dim3 gridDim( (dimension+2)/(NUMTHREADSPERBLOCK/2), (dimension+2)/(NUMTHREADSPERBLOCK/2) ); //-2 in /2
  dim3 blockDim(NUMTHREADSPERBLOCK , NUMTHREADSPERBLOCK);

  for( it = 0; it < iterations; ++it ){
  
  evolve<<< gridDim, blockDim  >>>( d_matrix, d_matrix_new, dimension );
  //evolve<<< (dimension+NUMTHREADSPERBLOCK)/NUMTHREADSPERBLOCK , NUMTHREADSPERBLOCK >>>( d_matrix, d_matrix_new, dimension );

    // swap the pointers
    tmp_matrix = d_matrix;
    d_matrix = d_matrix_new;
    d_matrix_new = tmp_matrix;
 }

  hipMemcpy( matrix, d_matrix, byte_dimension, hipMemcpyDeviceToHost );
  t_end = seconds();

  printf("final matrix\n");
  PRINT_MAT(dimension+2,dimension+2,matrix);

  printf( "\nelapsed time = %f seconds\n", t_end - t_start );
  printf( "\nmatrix[%zu,%zu] = %f\n", row_peek, col_peek, matrix[ ( row_peek + 1 ) * ( dimension + 2 ) + ( col_peek + 1 ) ] );

  save_gnuplot( matrix, dimension );
  

  free( matrix );
  free( matrix_new );
  hipFree(d_matrix);
  hipFree(d_matrix_new);


  return 0;

}

void PRINT_MAT(int P, int M, double * matr){
  for(int j = 0; j < P; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%0.1f ",matr[i+j*M]);
    }
    printf("\n");
  }
}

__global__ void evolve( double * matrix, double * matrix_new, int dimension ){

  __shared__ double shared_matrix[NUMTHREADSPERBLOCK*NUMTHREADSPERBLOCK];

  //int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  //int idx = threadIdx.x + (blockIdx.x * (blockDim.x/(NUMTHREADSPERBLOCK/2) );
  //int valx = (blockIdx.x * (blockDim.x/(NUMTHREADSPERBLOCK/2) );
  int idx = threadIdx.x + (blockIdx.x * (blockDim.x-2));
  int idy = threadIdx.y + (blockIdx.y * (blockDim.y-2));
  int valx = (blockIdx.x * (blockDim.x-2) );
  int valy = (blockIdx.y * (blockDim.y-2) );
  int i = threadIdx.x; //from 0 to NUMTHREADSPERBLOCK-1
  int j = threadIdx.y;

   if(idx <=(dimension + 1) && idy <=(dimension + 1) ){ //+1 perche prendo anche gli elem del bordo
    shared_matrix [ i *(NUMTHREADSPERBLOCK) + j ] = matrix[ (idx*(dimension + 2)) +(idy) ]; //+2 messo
    //printf("i= %i, j= %i; con idx=%i, idy=%i, shared=%f; mat=%f \n", i, j, idx, idy, shared_matrix [ i *(NUMTHREADSPERBLOCK) + j ], matrix[ idx*(dimension +2)+idy ]);
    }
  
  __syncthreads();


  if (i>=1 && j>=1 && i<NUMTHREADSPERBLOCK-1 && j<=NUMTHREADSPERBLOCK-1){ //qui non prendo i bordi, lavoro solo sugli elementi interni	
    if (idx>0 && idx<=dimension && idy>0 && idy<=dimension){
    if (idx> valx && idx<valx+NUMTHREADSPERBLOCK-1 && idy>valy && idy<valy + NUMTHREADSPERBLOCK-1){
    matrix_new[ (idx * (dimension+2)) + (idy) ] = ( 0.25 ) * 
    ( shared_matrix[ ( ( i-1 ) * ( NUMTHREADSPERBLOCK) ) + j ] +  //+2 tolto
    shared_matrix[ ( i * ( NUMTHREADSPERBLOCK ) ) + ( j+1 ) ] + 	  
    shared_matrix[ ( ( i+1 ) * ( NUMTHREADSPERBLOCK ) ) + j ] +
    shared_matrix[ ( i * ( NUMTHREADSPERBLOCK ) ) + ( j-1 ) ] );
    //printf("elem=(%i,%i);  intorno: (%i,%i)  (%i,%i)  (%i,%i)  (%i,%i);  matn =%f \n", idx, idy, i-1, j, i, j+1, i+1, j, i, j-1, matrix_new[ idx * (dimension+2) + idy ]  );
//printf("mat=%f, shared_matrix=%f \n", matrix[ ( ( idx-1 ) * (dimension + 2) ) + idy ], shared_matrix[ ( ( i-1 ) * ( NUMTHREADSPERBLOCK) ) + j ] );

  /* if(idx>0 && idx<=dimension){
    if(idy>0 && idy<=dimension){
      matrix_new[ idx * (dimension+2) + idy ] = ( 0.25 ) * 
	( matrix[ ( ( idx-1 ) * (dimension + 2) ) + idy ] + 
	  matrix[ ( idx * ( dimension +2 ) ) + ( idy+1 ) ] + 	  
	  matrix[ ( ( idx+1 ) * ( dimension + 2 ) ) + idy ] +
	  matrix[ ( idx * ( dimension + 2 ) ) + ( idy-1 ) ] );
	  //printf("elem=(%i,%i);  intorno: (%i,%i)  (%i,%i)  (%i,%i)  (%i,%i);  matn =%f \n", idx, idy, i-1, j, i, j+1, i+1, j, i, j-1, matrix_new[ idx * (dimension+2) + idy ]  );*/
    }
  }
}

  
}

void save_gnuplot( double *M, size_t dimension ){  

  size_t i , j;

  const double h = 0.1;

  FILE *file;

  file = fopen( "solution.dat", "w" );

  for( i = 0; i < dimension + 2; ++i )

    for( j = 0; j < dimension + 2; ++j )

      fprintf(file, "%f\t%f\t%f\n", h * j, -h * i, M[ ( i * ( dimension + 2 ) ) + j ] );

  fclose( file );

}



// A Simple timer for measuring the walltime

double seconds(){

    struct timeval tmp;
    double sec;
    gettimeofday( &tmp, (struct timezone *)0 );
    sec = tmp.tv_sec + ((double)tmp.tv_usec)/1000000.0;
    return sec;

}
