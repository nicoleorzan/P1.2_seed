
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

#define NUMTHREADSPERBLOCK 16

/*** function declarations ***/

void PRINT_MAT(int P, int M, double * matr);

// save matrix to file
void save_gnuplot( double *M, size_t dim );

// evolve Jacobi
__global__  void evolve( double * matrix, double *matrix_new, int dimension );


// return the elapsed time
double seconds( void );

/*** end function declaration ***/




int main(int argc, char* argv[]){

  // timing variables
  double t_start, t_end, increment;

  // indexes for loops
  size_t i, j, it;

  // initialize matrix

  double *matrix, *matrix_new, *tmp_matrix;
  double *d_matrix, *d_matrix_new;
  int dimension = 0, iterations = 0, row_peek = 0, col_peek = 0;
  size_t byte_dimension = 0;

 // check on input parameters

  if(argc != 5) {
    fprintf(stderr,"\nwrong number of arguments. Usage: ./a.out dim it n m\n");
    return 1;
  }

  dimension = atoi(argv[1]);
  iterations = atoi(argv[2]);
  row_peek = atoi(argv[3]);
  col_peek = atoi(argv[4]);

  printf("matrix size = %zu\n", dimension);
  printf("number of iterations = %zu\n", iterations);
  printf("element for checking = Mat[%zu,%zu]\n",row_peek, col_peek);


  if((row_peek > dimension) || (col_peek > dimension)){
    fprintf(stderr, "Cannot Peek a matrix element outside of the matrix dimension\n");
    fprintf(stderr, "Arguments n and m must be smaller than %zu\n", dimension);
    return 1;
  }


  byte_dimension = sizeof(double*) * ( dimension + 2 ) * ( dimension + 2 );
  matrix = ( double* )malloc( byte_dimension );
  matrix_new = ( double* )malloc( byte_dimension );

  memset( matrix, 0, byte_dimension );
  memset( matrix_new, 0, byte_dimension );

  hipMalloc((void**) &d_matrix, byte_dimension ); //allocating space for d_matrix
  hipMalloc((void**) &d_matrix_new, byte_dimension );

  //fill initial values  

  for( i = 1; i <= dimension; ++i ){
    for( j = 1; j <= dimension; ++j ){
      matrix[ ( i * ( dimension + 2 ) ) + j ] = 0.5;
    }
  }
	     
  // set up borders 
  increment = 100.0 / ( dimension + 1 );

  
 for( i=1; i <= dimension+1; ++i ){
    matrix[ i * ( dimension + 2 ) ] = i * increment;
    matrix[ ( ( dimension + 1 ) * ( dimension + 2 ) ) + ( dimension + 1 - i ) ] = i * increment;
    matrix_new[ i * ( dimension + 2 ) ] = i * increment;
    matrix_new[ ( ( dimension + 1 ) * ( dimension + 2 ) ) + ( dimension + 1 - i ) ] = i * increment;
  }

  printf("initial matrix\n");
  // PRINT_MAT(dimension+2,dimension+2,matrix);

  // start algorithm
  t_start = seconds();

  hipMemcpy( d_matrix, matrix, byte_dimension, hipMemcpyHostToDevice );
  hipMemcpy( d_matrix_new, matrix_new, byte_dimension, hipMemcpyHostToDevice );

  dim3 gridDim( (dimension+NUMTHREADSPERBLOCK)/NUMTHREADSPERBLOCK, (dimension+NUMTHREADSPERBLOCK)/NUMTHREADSPERBLOCK );
  dim3 blockDim(NUMTHREADSPERBLOCK , NUMTHREADSPERBLOCK);

  for( it = 0; it < iterations; ++it ){
  
  evolve<<< gridDim, blockDim  >>>( d_matrix, d_matrix_new, dimension );
    //evolve<<< (dimension+NUMTHREADSPERBLOCK)/NUMTHREADSPERBLOCK , NUMTHREADSPERBLOCK >>>( d_matrix, d_matrix_new, dimension );

    // swap the pointers
    tmp_matrix = d_matrix;
    d_matrix = d_matrix_new;
    d_matrix_new = tmp_matrix;
 }

  hipMemcpy( matrix, d_matrix, byte_dimension, hipMemcpyDeviceToHost );
  t_end = seconds();

  printf("final matrix\n");
  //PRINT_MAT(dimension+2,dimension+2,matrix);

  printf( "\nelapsed time = %f seconds\n", t_end - t_start );
  printf( "\nmatrix[%zu,%zu] = %f\n", row_peek, col_peek, matrix[ ( row_peek + 1 ) * ( dimension + 2 ) + ( col_peek + 1 ) ] );

  save_gnuplot( matrix, dimension );
  

  free( matrix );
  free( matrix_new );
  hipFree(d_matrix);
  hipFree(d_matrix_new);


  return 0;

}

void PRINT_MAT(int P, int M, double * matr){
  for(int j = 0; j < P; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%0.1f ",matr[i+j*M]);
    }
    printf("\n");
  }
}

__global__ void evolve( double * matrix, double * matrix_new, int dimension ){

  /*  __shared__ double * shared_matrix; //pointer to the part of the matrix I want to share

  shared_matrix = ( double* )malloc( NUMTHREADSPERBLOCK * NUMTHREADSPERBLOCK );
  */
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  int idy = threadIdx.y + (blockIdx.y * blockDim.y); 
  
  /* if (idx < dimension + 2){
    if(idy < dimension + 2){
      shared_matrix [ idx*(dimension)+idy ] = matrix[ idx*(dimension)+idy ];
    }
  }
  __syncthreads();*/
  
  if (idx>0 && idx<=dimension){
    if(idy>0 && idy<=dimension){
      matrix_new[ ( idx * ( dimension + 2 ) ) + idy ] = ( 0.25 ) * 
	( matrix[ ( ( idx - 1 ) * ( dimension + 2 ) ) + idy ] + 
	  matrix[ ( idx * ( dimension + 2 ) ) + ( idy + 1 ) ] + 	  
	  matrix[ ( ( idx + 1 ) * ( dimension + 2 ) ) + idy ] + 
	  matrix[ ( idx * ( dimension + 2 ) ) + ( idy - 1 ) ] ); 
    }
  }
}

void save_gnuplot( double *M, size_t dimension ){  

  size_t i , j;

  const double h = 0.1;

  FILE *file;

  file = fopen( "solution.dat", "w" );

  for( i = 0; i < dimension + 2; ++i )

    for( j = 0; j < dimension + 2; ++j )

      fprintf(file, "%f\t%f\t%f\n", h * j, -h * i, M[ ( i * ( dimension + 2 ) ) + j ] );

  fclose( file );

}



// A Simple timer for measuring the walltime

double seconds(){

    struct timeval tmp;
    double sec;
    gettimeofday( &tmp, (struct timezone *)0 );
    sec = tmp.tv_sec + ((double)tmp.tv_usec)/1000000.0;
    return sec;

}
