
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<assert.h>

#define N 4
#define THREADSPERBLOCK 4

__global__ void matmult( double * d_A, double * d_B, double * d_C ){

  __shared__ double Arow[N];

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx<N*N){
    Arow[threadIdx.x]=d_A[idx];
  }
  __syncthreads();

  for(int  k = 0; k < N; k++ ) {
     d_C[idx] += Arow[k] *d_B[k * N + threadIdx.x];
   }

}

void PRINT_MAT(int P, int M, double * matr){
  for(int j = 0; j < P; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%f ",matr[i+j*M]);
     }
    printf("\n");
  }
}


int main(){
  
  double * h_A , * h_B, * h_C;
  double * d_A , * d_B, * d_C;
  size_t matsize = N * N * sizeof(double); //long integer
  
  h_A = (double *) malloc( matsize );
  h_B = (double *) malloc( matsize );
  h_C = (double *) malloc( matsize );
  
  hipMalloc((void**) &d_A, matsize );
  hipMalloc((void**) &d_B, matsize );
  hipMalloc((void**) &d_C, matsize );
  
  for(int i=0;i<N*N;i++){
    h_A[i]=( rand() % 100 + 1 );// (double )i;
    h_B[i]=( rand() % 100 + 1 ); //(double )i;
    h_C[i]=0.;
  }
  
  printf("matrice A:\n");  
  PRINT_MAT(N,N,h_A);
  printf("matrice B:\n");
  PRINT_MAT(N,N,h_B);

  hipMemcpy( d_A, h_A, matsize, hipMemcpyHostToDevice );
  hipMemcpy( d_B, h_B, matsize, hipMemcpyHostToDevice );
  hipMemcpy( d_C, h_C, matsize, hipMemcpyHostToDevice );

  dim3 blockDim(THREADSPERBLOCK, THREADSPERBLOCK);
  dim3 gridDim((N*N)/THREADSPERBLOCK, (N*N)/THREADSPERBLOCK);
  matmult<<< gridDim, blockDim >>>( d_A, d_B, d_C);
  hipMemcpy( h_C, d_C, matsize, hipMemcpyDeviceToHost );
  
  printf("matrice C=A*B:\n");
  PRINT_MAT(N,N,h_C);
  
  free(h_A);
  free(h_B);
  free(h_C);
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  
  return 0;
  
}
