
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<assert.h>

#define N 4
#define THREADSPERBLOCK 4

__global__ void matmult( double * d_A, double * d_B, double * d_C){

  int i = threadIdx.x + (blockIdx.x * blockDim.x);
  int j = threadIdx.y + (blockIdx.y * blockDim.y);
 
  if (i<N && j<N){
    for (int k=0; k<N; k++){
      d_C[j+i*N]+=d_A[i*N+k]*d_B[k*N+j];
    }
  }


}

void PRINT_MAT(int P, int M, double * matr){
  for(int j = 0; j < P; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%f ",matr[i+j*M]);
    }
    printf("\n");
  }
}

int main(){
  
  double * h_A, * h_B, * h_C;
  double * d_A, * d_B, * d_C;
  size_t matsize = N * N * sizeof(double);

  h_A = (double *) malloc( matsize );
  h_B = (double *) malloc( matsize );
  h_C = (double *) malloc( matsize );
  
  hipMalloc((void**) &d_A, matsize );
  hipMalloc((void**) &d_B, matsize );
  hipMalloc((void**) &d_C, matsize );

 for(int i=0;i<N*N;i++){
    h_A[i]=( rand() % 100 + 1 ); //(double )i;
    h_B[i]=( rand() % 100 + 1 ); //(double )i;
    h_C[i]=0.;
  }
  printf("matrice A:\n");  
  PRINT_MAT(N,N,h_A);
  printf("matrice B:\n");
  PRINT_MAT(N,N,h_B);

  hipMemcpy( d_A, h_A, matsize, hipMemcpyHostToDevice );
  hipMemcpy( d_B, h_B, matsize, hipMemcpyHostToDevice );
  hipMemcpy( d_C, h_C, matsize, hipMemcpyHostToDevice );

  dim3 blockDim(THREADSPERBLOCK, THREADSPERBLOCK);
  dim3 gridDim(N/THREADSPERBLOCK, N/THREADSPERBLOCK);
  matmult<<< gridDim, blockDim >>>( d_A, d_B, d_C);
  hipMemcpy( h_C, d_C, matsize, hipMemcpyDeviceToHost );
  
  printf("matrice C=A*B:\n");
  PRINT_MAT(N,N,h_C);
  
  free(h_A);
  free(h_B);
  free(h_C);
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
return 0;
}
