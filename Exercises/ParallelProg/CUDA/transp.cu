
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<assert.h>

#define N 4 //size of the matrix in one dimension
#define THREADSPERBLOCK 4

void PRINT_MAT(int P, int M, double * matr){
  for(int j = 0; j < P; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%f ",matr[i+j*M]);
    }
    printf("\n");
  }
}

__global__ void transpose( double * d_A, double * d_AT ){
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  int i = idx/N;
  int j = idx%N;
  
  if (idx<N*N)  d_AT[i+j*N] = d_A[j+i*N];
}
  
  
  
  int main(){
    
    double * h_A, * h_AT;
    double * d_A, * d_AT;
    size_t matsize = N * N * sizeof(double); //long integer
    int nblocks = ((N*N)+THREADSPERBLOCK)/THREADSPERBLOCK;
    
    h_A = (double *) malloc( matsize );
    h_AT = (double *) malloc( matsize );
    
    hipMalloc((void**) &d_A, matsize );
    hipMalloc((void**) &d_AT, matsize );
    
    for(int i=0;i<N*N;i++){
      h_A[i]=(double )i;
      h_AT[i]=0.;
    }
    
    printf("initial matrix:\n");
    PRINT_MAT(N,N,h_A);
    
    hipMemcpy( d_A, h_A, matsize, hipMemcpyHostToDevice );
    hipMemcpy( d_AT, h_AT, matsize, hipMemcpyHostToDevice );
    transpose<<< nblocks, THREADSPERBLOCK >>>( d_A , d_AT );
    hipMemcpy( h_AT, d_AT, matsize, hipMemcpyDeviceToHost );

    printf("transpose matrix:\n");
    PRINT_MAT(N,N,h_AT);
    
    free(h_A);
    free(h_AT);
    
    hipFree(d_A);
    hipFree(d_AT);
    
    return 0;
    
  }
