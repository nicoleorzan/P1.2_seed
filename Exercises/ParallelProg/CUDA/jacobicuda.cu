
#include <hip/hip_runtime.h>
#include <string.h>

#include <stdlib.h>

#include <stdio.h>

#include <time.h>

#include <sys/time.h>

#include<assert.h>

#define N 4
#define NUMTHREADSPERBLOCK 4

/*** function declarations ***/

void PRINT_MAT(int P, int M, double * matr){
  for(int j = 0; j < P; j++ ){
    for(int i = 0; i < M; i++ ){
      printf("%0.1f ",matr[i+j*M]);
    }
    printf("\n");
  }
}

// save matrix to file
void save_gnuplot( double *M, size_t dim );

// evolve Jacobi
__global__  void evolve( double * matrix, double *matrix_new, size_t dimension );


// return the elapsed time
double seconds( void );

/*** end function declaration ***/




int main(int argc, char* argv[]){

  // timing variables
  double t_start, t_end, increment;

  // indexes for loops
  size_t i, j, it;

  // initialize matrix

  double *matrix, *matrix_new, *tmp_matrix;
  double *d_matrix, *d_matrix_new;//, *tmp_matrix;
  size_t dimension = 0, iterations = 0, row_peek = 0, col_peek = 0;
  size_t byte_dimension = 0;

 // check on input parameters

  if(argc != 5) {
    fprintf(stderr,"\nwrong number of arguments. Usage: ./a.out dim it n m\n");
    return 1;
  }

  dimension = atoi(argv[1]);
  iterations = atoi(argv[2]);
  row_peek = atoi(argv[3]);
  col_peek = atoi(argv[4]);

  printf("matrix size = %zu\n", dimension);
  printf("number of iterations = %zu\n", iterations);
  printf("element for checking = Mat[%zu,%zu]\n",row_peek, col_peek);


  if((row_peek > dimension) || (col_peek > dimension)){
    fprintf(stderr, "Cannot Peek a matrix element outside of the matrix dimension\n");
    fprintf(stderr, "Arguments n and m must be smaller than %zu\n", dimension);
    return 1;
  }


  byte_dimension = sizeof(double*) * ( dimension + 2 ) * ( dimension + 2 );
  matrix = ( double* )malloc( byte_dimension );
  matrix_new = ( double* )malloc( byte_dimension );

  memset( matrix, 0, byte_dimension );
  memset( matrix_new, 0, byte_dimension );

  hipMalloc((void**) &d_matrix, byte_dimension );
  hipMalloc((void**) &d_matrix_new, byte_dimension );

  //fill initial values  

  int num=0;
  for( i = 1; i <= dimension; ++i ){
    for( j = 1; j <= dimension; ++j ){
      matrix[ ( i * ( dimension + 2 ) ) + j ] = 0.5;
      num++;
    }
  }
  printf("num=%i\n",num);
	     
  // set up borders 
  increment = 100.0 / ( dimension + 1 );

  
 for( i=1; i <= dimension+1; ++i ){
    matrix[ i * ( dimension + 2 ) ] = i * increment;
    matrix[ ( ( dimension + 1 ) * ( dimension + 2 ) ) + ( dimension + 1 - i ) ] = i * increment;
    matrix_new[ i * ( dimension + 2 ) ] = i * increment;
    matrix_new[ ( ( dimension + 1 ) * ( dimension + 2 ) ) + ( dimension + 1 - i ) ] = i * increment;
  }

  printf("initial matrix\n");
  PRINT_MAT(dimension+2,dimension+2,matrix);

  hipMemcpy( d_matrix, matrix, byte_dimension, hipMemcpyHostToDevice );
  hipMemcpy( d_matrix_new, matrix_new, byte_dimension, hipMemcpyHostToDevice );

  // start algorithm
  t_start = seconds();

  for( it = 0; it < iterations; ++it ){
  
  // evolve<<< 6,6 >>>( d_matrix, d_matrix_new, dimension );
  evolve<<< (((dimension+2)*(dimension+2))+NUMTHREADSPERBLOCK)/NUMTHREADSPERBLOCK , NUMTHREADSPERBLOCK >>>( d_matrix, d_matrix_new, dimension );

    // swap the pointers
    tmp_matrix = d_matrix;
    d_matrix = d_matrix_new;
    d_matrix_new = tmp_matrix;
 }

  t_end = seconds();
  hipMemcpy( matrix, d_matrix, byte_dimension, hipMemcpyDeviceToHost );

  printf("\n");
  printf("final matrix\n");
  PRINT_MAT(dimension+2,dimension+2,matrix);

  printf( "\nelapsed time = %f seconds\n", t_end - t_start );
  printf( "\nmatrix[%zu,%zu] = %f\n", row_peek, col_peek, matrix[ ( row_peek + 1 ) * ( dimension + 2 ) + ( col_peek + 1 ) ] );

  save_gnuplot( matrix, dimension );
  

  free( matrix );
  free( matrix_new );
  hipFree(d_matrix);
  hipFree(d_matrix_new);


  return 0;

}



__global__ void evolve( double * matrix, double * matrix_new, size_t dimension ){

  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  int i = idx/(dimension+2);
  int j = idx%(dimension+2);
 

       if (i>0 && i<=dimension){
        if(j>0 && j<=dimension){
	matrix_new[ ( i * ( dimension + 2 ) ) + j ] = ( 0.25 ) * 
	( matrix[ ( ( i - 1 ) * ( dimension + 2 ) ) + j ] + 
	  matrix[ ( i * ( dimension + 2 ) ) + ( j + 1 ) ] + 	  
	  matrix[ ( ( i + 1 ) * ( dimension + 2 ) ) + j ] + 
	  matrix[ ( i * ( dimension + 2 ) ) + ( j - 1 ) ] ); 
}
}
}

void save_gnuplot( double *M, size_t dimension ){  

  size_t i , j;

  const double h = 0.1;

  FILE *file;

  file = fopen( "solution.dat", "w" );

  for( i = 0; i < dimension + 2; ++i )

    for( j = 0; j < dimension + 2; ++j )

      fprintf(file, "%f\t%f\t%f\n", h * j, -h * i, M[ ( i * ( dimension + 2 ) ) + j ] );

  fclose( file );

}



// A Simple timer for measuring the walltime

double seconds(){

    struct timeval tmp;
    double sec;
    gettimeofday( &tmp, (struct timezone *)0 );
    sec = tmp.tv_sec + ((double)tmp.tv_usec)/1000000.0;
    return sec;

}